
#include <hip/hip_runtime.h>
__global__ void kernelC3(double *A,double *B,double *C,int width, double r){
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    int step;
    double prod_val = 0;
    if((idy>=(int)(width*(1-r)))||(idx>=(int)(width*r))) return;
    
    for(step=0;step<width;step++){
        prod_val += A[idy*width+step] * B[step*(int)(width*r)+idx];
    }
    
    
    C[idy*(int)(width*r)+idx] = prod_val;
}